#include "hip/hip_runtime.h"

//librer�a glfw

#include <GL/glew.h>

// Include GLFW
#include <GLFW/glfw3.h>
GLFWwindow* window;

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>
#include <>

#include<stdio.h>


static void HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}


#define     WIDTH    800
#define		HEIGHT	 600
#define EPSILON			0.005f


GLuint  bufferObj;
hipGraphicsResource *resource;

int cambiofx = 1;
int cambiofy = 1;


#define NSPHERES 3
#define NPLANES 1
#define NLIGHTS 2


//spheres Code 25
#define SPHERE 25
__constant__ float sphere[NSPHERES * 4];  //NSPHERES spheres will be stored as cx,cy,cz,r.
__constant__ float sphereColor[NSPHERES * 4]; //NSPHERES colors will be stored as r,g,b,refraction.
__constant__ float sphereLightProperties[NSPHERES * 2]; //NSPHERES properties will be stored as diff, reflection.

//planes Code 26
#define PLANE 26
__constant__ float plane[NPLANES * 4]; //NPLANES plane will be stored as Nx Ny Nz D.
__constant__ float planeColor[NPLANES * 3]; //NPLANES color will be stored as r,g,b.
__constant__ float planeLightProperties[NPLANES * 2]; //NPLANES properties will be stored as diff, reflection.

//lights Code 27
#define LIGHT 27
__constant__ float light[NLIGHTS * 4]; //NLIGHTS light will be stored as cx, cy, cz,r.
__constant__ float lightColor[NLIGHTS * 3]; //NLIGHTS light color will be stored as r,g,b.


//__constant__ float transformation[16];

__device__ void intersections(float originX, float originY, float originZ,
	float dX, float dY, float dZ,
	char* primitive, int* position, float* distance){
	*distance = 1000000.0f; //inf
	*primitive = -1;
	*position = -1;

	float vX, vY, vZ;
	float discriminant;
	float t;

	//check spheres
	for (int i = 0; i<NSPHERES; i++){
		vX = originX - sphere[i * 4];
		vY = originY - sphere[i * 4 + 1];
		vZ = originZ - sphere[i * 4 + 2];

		discriminant = (vX*dX + vY*dY + vZ*dZ)*(vX*dX + vY*dY + vZ*dZ) - (vX*vX + vY*vY + vZ*vZ) + (sphere[i * 4 + 3] * sphere[i * 4 + 3]); //dot(v,d)^2 - (dot(v,v) - r^2)

		if (discriminant > 0){
			t = sqrtf(discriminant);
			t = fminf(-1.0f*(vX*dX + vY*dY + vZ*dZ) + t, -1.0f*(vX*dX + vY*dY + vZ*dZ) - t);
			if (t > 0){
				*distance = fminf(t, *distance);
				if (*distance == t){ //found a closer primitive
					*primitive = SPHERE;
					*position = i;
				}
			}
		}
	}


	
	for (int i = 0; i<NPLANES; i++){
		t = (plane[i * 4] * dX + plane[i * 4 + 1] * dY + plane[i * 4 + 2] * dZ);
		if (t != 0){
			t=-(plane[i * 4] * originX + plane[i * 4 + 1] * originY + plane[i * 4 + 2] * originZ + plane[i * 4 + 3])
				/t ;
			if (t > 0){
				*distance = fminf(t, *distance);
				if (t == *distance){ //found a closer primitive
					*primitive = PLANE;
					*position = i;
				}
			}
		}
	}


	//check lights
	for (int i = 0; i<NLIGHTS; i++){
		vX = originX - light[i * 4];
		vY = originY - light[i * 4 + 1];
		vZ = originZ - light[i * 4 + 2];

		discriminant = (vX*dX + vY*dY + vZ*dZ)*(vX*dX + vY*dY + vZ*dZ) - (vX*vX + vY*vY + vZ*vZ) + (light[i * 4 + 3] * light[i * 4 + 3]); //dot(v,d)^2 - (dot(v,v) - r^2))

		if (discriminant >= 0){
			t = fminf(-1.0f*(vX*dX + vY*dY + vZ*dZ) + sqrtf(discriminant), -1.0f*(vX*dX + vY*dY + vZ*dZ) - sqrtf(discriminant));
			if (t >=0){
				*distance = fminf(t, *distance);
				if (*distance == t){ //found a closer primitive
					*primitive = LIGHT;
					*position = i;
				}
			}
		}
	}

}


__device__ void newRay2(float originX, float originY, float originZ, float vX, float vY, float vZ, float &r, float &g, float &b){
	float intersectionPointX, intersectionPointY, intersectionPointZ;
	float nX, nY, nZ, lX, lY, lZ;
	float rX, rY, rZ;
	float distance;
	float shade = 0.0f;
	float aux;
	char primitiveType;
	int primitivePosition;

	float distanceR;
	char primitiveTypeR;
	int primitivePositionR;

	//Normalize V
	aux = 1 / sqrtf(vX*vX + vY*vY + vZ*vZ);
	vX *= aux;
	vY *= aux;
	vZ *= aux;

	intersections(originX, originY, originZ, vX, vY, vZ, &primitiveType, &primitivePosition, &distance);

	r = 0.0f;
	g = 0.0f;
	b = 0.0f;
	if (primitiveType == LIGHT){ //if it hit a light, assign the color and stop this ray
		r += lightColor[3 * primitivePosition];
		g += lightColor[3 * primitivePosition + 1];
		b += lightColor[3 * primitivePosition + 2];
		//	isActive=false;
	}
	else if (primitiveType != -1 && distance>0){

		intersectionPointX = originX + distance*vX;
		intersectionPointY = originY + distance*vY;
		intersectionPointZ = originZ + distance*vZ;

		if (primitiveType == SPHERE){
			nX = intersectionPointX - sphere[4 * primitivePosition];
			nY = intersectionPointY - sphere[4 * primitivePosition + 1];
			nZ = intersectionPointZ - sphere[4 * primitivePosition + 2];
			//Normalize n
			aux = 1.0f / sqrtf(nX*nX + nY*nY + nZ*nZ);
			nX *= aux;
			nY *= aux;
			nZ *= aux;

		}
		else if (primitiveType == PLANE){
			nX = plane[4 * primitivePosition];
			nY = plane[4 * primitivePosition + 1];
			nZ = plane[4 * primitivePosition + 2];
		}



		for (int l = 0; l<NLIGHTS; l++){

			//calculate light vector
			lX = light[4 * l] - intersectionPointX;
			lY = light[4 * l + 1] - intersectionPointY;
			lZ = light[4 * l + 2] - intersectionPointZ;
			//Normalize l
			aux = 1.0f / sqrtf(lX*lX + lY*lY + lZ*lZ);
			lX *= aux;
			lY *= aux;
			lZ *= aux;

			if (primitiveType == SPHERE){
				//Calculate diffuse shading
				if (sphereLightProperties[primitivePosition * 2]>0){
					aux = lX*nX + lY*nY + lZ*nZ; //dot(n,l)
					if (aux>0){
						aux *= sphereLightProperties[primitivePosition * 2];
						r += aux*sphereColor[primitivePosition * 4] * lightColor[l * 3];
						g += aux*sphereColor[primitivePosition * 4 + 1] * lightColor[l * 3 + 1];
						b += aux*sphereColor[primitivePosition * 4 + 2] * lightColor[l * 3 + 2];
					}
					////Calculate specular component
					if (1.0f - sphereLightProperties[primitivePosition * 2]>0){
						//r=l-2dot(l,n)n
						rX = lX - 2.0f*(lX*nX + lY*nY + lZ*nZ)*nX;
						rY = lY - 2.0f*(lX*nX + lY*nY + lZ*nZ)*nY;
						rZ = lZ - 2.0f*(lX*nX + lY*nY + lZ*nZ)*nZ;

						aux = vX*rX + vY*rY + vZ*rZ;//dot(v,r)
						if (aux>0){
							aux *= powf(aux, 20)*(1.0f - sphereLightProperties[primitivePosition * 2]);
							r += aux*lightColor[l * 3];
							g += aux*lightColor[l * 3 + 1];
							b += aux*lightColor[l * 3 + 2];


						}
					}

				}

			}
			else if (primitiveType == PLANE){
				//Calculate diffuse shading
				if (planeLightProperties[primitivePosition * 2]>0){

					aux = lX*nX + lY*nY + lZ*nZ; //dot(l,n)
					if (aux>0){
						aux *= planeLightProperties[primitivePosition * 2];
						r += aux*planeColor[primitivePosition * 3] * lightColor[l * 3];
						g += aux*planeColor[primitivePosition * 3 + 1] * lightColor[l * 3 + 1];
						b += aux*planeColor[primitivePosition * 3 + 2] * lightColor[l * 3 + 2];
					}
				}

				////Calculate specular component
				if (1.0f - planeLightProperties[primitivePosition * 2]>0){
					//r=l-2dot(l,n)n
					rX = lX - 2.0f*(lX*nX + lY*nY + lZ*nZ)*nX;
					rY = lY - 2.0f*(lX*nX + lY*nY + lZ*nZ)*nY;
					rZ = lZ - 2.0f*(lX*nX + lY*nY + lZ*nZ)*nZ;

					aux = vX*rX + vY*rY + vZ*rZ;//dot(v,r)
					if (aux>0){
						aux *= powf(aux, 20)*(1.0f - planeLightProperties[primitivePosition * 2]);
						r += aux*lightColor[l * 3];
						g += aux*lightColor[l * 3 + 1];
						b += aux*lightColor[l * 3 + 2];


					}
				}

			}

			////Calculate shade
			shade = 0.5f;

			intersections(intersectionPointX + EPSILON*lX, intersectionPointY + EPSILON*lY, intersectionPointZ + EPSILON*lZ, lX, lY, lZ, &primitiveTypeR, &primitivePositionR, &distanceR);
			if (primitiveTypeR == LIGHT){
				shade = 1.0f;
			}

			r *= shade;
			g *= shade;
			b *= shade;

		}

	}

}


__device__ void newRay(float originX, float originY, float originZ, float vX, float vY, float vZ, float &r, float &g, float &b){
	float intersectionPointX, intersectionPointY, intersectionPointZ;
	float nX, nY, nZ, lX, lY, lZ;
	float rX, rY, rZ;
	float distance;
	float shade = 0.0f;
	float aux;
	char primitiveType;
	int primitivePosition;

	float distanceR;
	char primitiveTypeR;
	int primitivePositionR;
	
	//Normalize V
	aux = 1 / sqrtf(vX*vX + vY*vY + vZ*vZ);
	vX *= aux;
	vY *= aux;
	vZ *= aux;

	intersections(originX, originY, originZ, vX, vY, vZ, &primitiveType, &primitivePosition, &distance);

	r = 0.0f;
	g = 0.0f;
	b = 0.0f;
	if (primitiveType == LIGHT){ //if it hit a light, assign the color and stop this ray
		r += lightColor[3 * primitivePosition];
		g += lightColor[3 * primitivePosition + 1];
		b += lightColor[3 * primitivePosition + 2];
		//	isActive=false;
	}
	else if (primitiveType != -1 && distance>0){

		intersectionPointX = originX + distance*vX;
		intersectionPointY = originY + distance*vY;
		intersectionPointZ = originZ + distance*vZ;

		if (primitiveType == SPHERE){
			nX = intersectionPointX - sphere[4 * primitivePosition];
			nY = intersectionPointY - sphere[4 * primitivePosition + 1];
			nZ = intersectionPointZ - sphere[4 * primitivePosition + 2];
			//Normalize n
			aux = 1.0f / sqrtf(nX*nX + nY*nY + nZ*nZ);
			nX *= aux;
			nY *= aux;
			nZ *= aux;

		}
		else if (primitiveType == PLANE){
			nX = plane[4 * primitivePosition];
			nY = plane[4 * primitivePosition + 1];
			nZ = plane[4 * primitivePosition + 2];
		}



		for (int l = 0; l<NLIGHTS; l++){

			//calculate light vector
			lX = light[4 * l] - intersectionPointX;
			lY = light[4 * l + 1] - intersectionPointY;
			lZ = light[4 * l + 2] - intersectionPointZ;
			//Normalize l
			aux = 1.0f / sqrtf(lX*lX + lY*lY + lZ*lZ);
			lX *= aux;
			lY *= aux;
			lZ *= aux;

			if (primitiveType == SPHERE){
				//Calculate diffuse shading
				if (sphereLightProperties[primitivePosition * 2]>0){
					aux = lX*nX + lY*nY + lZ*nZ; //dot(n,l)
					if (aux>0){
						aux *= sphereLightProperties[primitivePosition * 2];
						r += aux*sphereColor[primitivePosition * 4] * lightColor[l * 3];
						g += aux*sphereColor[primitivePosition * 4 + 1] * lightColor[l * 3 + 1];
						b += aux*sphereColor[primitivePosition * 4 + 2] * lightColor[l * 3 + 2];
					}
					////Calculate specular component
					if (1.0f - sphereLightProperties[primitivePosition * 2]>0){
						//r=l-2dot(l,n)n
						rX = lX - 2.0f*(lX*nX + lY*nY + lZ*nZ)*nX;
						rY = lY - 2.0f*(lX*nX + lY*nY + lZ*nZ)*nY;
						rZ = lZ - 2.0f*(lX*nX + lY*nY + lZ*nZ)*nZ;

						aux = vX*rX + vY*rY + vZ*rZ;//dot(v,r)
						if (aux>0){
							aux *= powf(aux, 20)*(1.0f - sphereLightProperties[primitivePosition * 2]);
							r += aux*lightColor[l * 3];
							g += aux*lightColor[l * 3 + 1];
							b += aux*lightColor[l * 3 + 2];


						}
					}


					//Calculate reflection
					if (sphereLightProperties[2 * primitivePosition + 1] > 0)
					{
						//r=v-2dot(v,n)n
						rX = vX - 2.0f*(vX*nX + vY*nY + vZ*nZ)*nX;
						rY = vY - 2.0f*(vX*nX + vY*nY + vZ*nZ)*nY;
						rZ = vZ - 2.0f*(vX*nX + vY*nY + vZ*nZ)*nZ;

						aux = sphereLightProperties[2 * primitivePosition + 1];
						float rRef, gRef, bRef;
						newRay2(intersectionPointX + EPSILON*rX, intersectionPointY + EPSILON*rY, intersectionPointZ + EPSILON*rZ, rX, rY, rZ, rRef, gRef, bRef);

						r += aux *rRef * sphereColor[primitivePosition * 4];
						g += aux*gRef * sphereColor[primitivePosition * 4 + 1];
						b += aux*bRef * sphereColor[primitivePosition * 4 + 2];
					}

				}

			}
			else if (primitiveType == PLANE){
				//Calculate diffuse shading
				if (planeLightProperties[primitivePosition * 2]>0){

					aux = lX*nX + lY*nY + lZ*nZ; //dot(l,n)
					if (aux>0){
						aux *= planeLightProperties[primitivePosition * 2];
						r += aux*planeColor[primitivePosition * 3] * lightColor[l * 3];
						g += aux*planeColor[primitivePosition * 3 + 1] * lightColor[l * 3 + 1];
						b += aux*planeColor[primitivePosition * 3 + 2] * lightColor[l * 3 + 2];
					}
				}

				////Calculate specular component
				if (1.0f - planeLightProperties[primitivePosition * 2]>0){
					//r=l-2dot(l,n)n
					rX = lX - 2.0f*(lX*nX + lY*nY + lZ*nZ)*nX;
					rY = lY - 2.0f*(lX*nX + lY*nY + lZ*nZ)*nY;
					rZ = lZ - 2.0f*(lX*nX + lY*nY + lZ*nZ)*nZ;

					aux = vX*rX + vY*rY + vZ*rZ;//dot(v,r)
					if (aux>0){
						aux *= powf(aux, 20)*(1.0f - planeLightProperties[primitivePosition * 2]);
						r += aux*lightColor[l * 3];
						g += aux*lightColor[l * 3 + 1];
						b += aux*lightColor[l * 3 + 2];


					}
				}


				//Calculate reflection
				if (planeLightProperties[2 * primitivePosition + 1] > 0)
				{
					//r=v-2dot(v,n)n
					rX = vX - 2.0f*(vX*nX + vY*nY + vZ*nZ)*nX;
					rY = vY - 2.0f*(vX*nX + vY*nY + vZ*nZ)*nY;
					rZ = vZ - 2.0f*(vX*nX + vY*nY + vZ*nZ)*nZ;

					aux = planeLightProperties[2 * primitivePosition + 1];
					float rRef, gRef, bRef;
					newRay2(intersectionPointX + EPSILON*rX, intersectionPointY + EPSILON*rY, intersectionPointZ + EPSILON*rZ, rX, rY, rZ, rRef, gRef, bRef);

					r += aux *rRef * planeColor[primitivePosition * 4];
					g += aux*gRef * planeColor[primitivePosition * 4 + 1];
					b += aux*bRef * planeColor[primitivePosition * 4 + 2];
				}

			}

			////Calculate shade
			shade = 0.5f;

			intersections(intersectionPointX + EPSILON*lX, intersectionPointY + EPSILON*lY, intersectionPointZ + EPSILON*lZ, lX, lY, lZ, &primitiveTypeR, &primitivePositionR, &distanceR);
			if (primitiveTypeR == LIGHT){
				shade = 1.0f;
			}

			r *= shade;
			g *= shade;
			b *= shade;



		}

	}


}

__global__ void rayTrace(float originX, float originY, float originZ, float xInf, float yInf, int width, int height, uchar4* output){

	float destX, destY, destZ;     //represents a pixel of the screen on a virtual plane
	float vX, vY, vZ; 		     //vector parallel to the traced ray
	float r = 0.0f, g = 0.0f, b = 0.0f;  //final color
	float intersectionPointX, intersectionPointY, intersectionPointZ;
	float nX, nY, nZ, lX, lY, lZ;
	float rX, rY, rZ;
	float distance;
	float shade=0.0f;
	float aux;
	char primitiveType;
	int primitivePosition;

	float distanceR;
	char primitiveTypeR;
	int primitivePositionR;

	//bool isActive;
	for (int globalTidY = threadIdx.y + blockDim.y*blockIdx.y; globalTidY<height; globalTidY += gridDim.y*blockDim.y){ //stride, in case threads
		for (int globalTidX = threadIdx.x + blockDim.x*blockIdx.x; globalTidX<width; globalTidX += gridDim.x*blockDim.x){//need to do more work

			//Initialize the world coordinate of the point of the screen that each thread will process.
			destX = xInf +globalTidX * 0.01f;
			destY = yInf + globalTidY * 0.01f;
			destZ = 0.0f;

			//We would transform the origin and dest vectors here using the transformation matrix. It's static for now.

			//initialize V that describes the line L(t)=origin+tV. Where origin is a vector that points to the center of the camera,
			//and V a vector parallel to the traced ray.
			vX = destX - originX;
			vY = destY - originY;
			vZ = destZ - originZ;

			//Normalize V
			aux = 1/sqrtf(vX*vX + vY*vY + vZ*vZ);
			vX *= aux;
			vY *= aux;
			vZ *= aux;
		
			intersections(originX, originY, originZ, vX, vY, vZ, &primitiveType, &primitivePosition, &distance);
			
			r = 0.0f;
			g = 0.0f;
			b = 0.0f;
			if (primitiveType == LIGHT){ //if it hit a light, assign the color and stop this ray
				r += lightColor[3 * primitivePosition];
				g += lightColor[3 * primitivePosition + 1];
				b += lightColor[3 * primitivePosition + 2];
				//	isActive=false;
			}
			else if (primitiveType != -1 && distance>0){

				intersectionPointX = originX + distance*vX;
				intersectionPointY = originY + distance*vY;
				intersectionPointZ = originZ + distance*vZ;

				if (primitiveType == SPHERE){
					nX = intersectionPointX - sphere[4 * primitivePosition];
					nY = intersectionPointY - sphere[4 * primitivePosition + 1];
					nZ = intersectionPointZ - sphere[4 * primitivePosition + 2];
					//Normalize n
					aux = 1.0f / sqrtf(nX*nX + nY*nY + nZ*nZ);
					nX *= aux;
					nY *= aux;
					nZ *= aux;

				}
				else if (primitiveType == PLANE){
					nX = plane[4 * primitivePosition];
					nY = plane[4 * primitivePosition + 1];
					nZ = plane[4 * primitivePosition + 2];
				}

				

				for (int l = 0; l<NLIGHTS; l++){
					//calculate light vector
					lX = light[4 * l] - intersectionPointX;
					lY = light[4 * l + 1] - intersectionPointY;
					lZ = light[4 * l + 2] - intersectionPointZ;
					//Normalize l
					aux = 1.0f/sqrtf(lX*lX + lY*lY + lZ*lZ);
					lX *=  aux;
					lY *=  aux;
					lZ *=  aux;

					/*if (globalTidX == 69 && globalTidY == 1){
						printf("light %d %f\n", l, distance);
						printf("ip(%f, %f, %f)\n", intersectionPointX, intersectionPointY, intersectionPointZ);
						printf("l(%f, %f, %f)\n", lX, lY, lZ);
					}*/
					if (primitiveType == SPHERE){
						//Calculate diffuse shading
						if (sphereLightProperties[primitivePosition * 2]>0){
							aux = lX*nX + lY*nY + lZ*nZ; //dot(n,l)
							if (aux>0){
								aux *= sphereLightProperties[primitivePosition * 2];
								r += aux*sphereColor[primitivePosition * 4] * lightColor[l * 3];
								g += aux*sphereColor[primitivePosition * 4 + 1] * lightColor[l * 3 + 1];
								b += aux*sphereColor[primitivePosition * 4 + 2] * lightColor[l * 3 + 2];
							}
							////Calculate specular component
							if (1.0f - sphereLightProperties[primitivePosition * 2]>0){
								//r=l-2dot(l,n)n
								rX = lX - 2.0f*(lX*nX + lY*nY + lZ*nZ)*nX;
								rY = lY - 2.0f*(lX*nX + lY*nY + lZ*nZ)*nY;
								rZ = lZ - 2.0f*(lX*nX + lY*nY + lZ*nZ)*nZ;

								aux = vX*rX + vY*rY + vZ*rZ;//dot(v,r)
								if (aux>0){
									aux *= powf(aux, 20)*(1.0f - sphereLightProperties[primitivePosition * 2]);
									r += aux*lightColor[l * 3];
									g += aux*lightColor[l * 3 + 1];
									b += aux*lightColor[l * 3 + 2];

									
								}
							}
								

							//Calculate reflection
							if (sphereLightProperties[2*primitivePosition+1] > 0)
							{
								//r=v-2dot(v,n)n
								rX = vX - 2.0f*(vX*nX + vY*nY + vZ*nZ)*nX;
								rY = vY - 2.0f*(vX*nX + vY*nY + vZ*nZ)*nY;
								rZ = vZ - 2.0f*(vX*nX + vY*nY + vZ*nZ)*nZ;
							
								aux = sphereLightProperties[2 * primitivePosition + 1];
								float rRef, gRef,bRef;
								newRay(intersectionPointX + EPSILON*rX, intersectionPointY + EPSILON*rY, intersectionPointZ + EPSILON*rZ, rX, rY, rZ, rRef, gRef, bRef);
								
								r+= aux *rRef * sphereColor[primitivePosition*4];
								g +=aux*gRef * sphereColor[primitivePosition * 4 + 1];
								b +=aux*bRef * sphereColor[primitivePosition * 4 + 2];
							}

						}

					}
					else if (primitiveType == PLANE){
						//Calculate diffuse shading
						if (planeLightProperties[primitivePosition * 2]>0){

							aux = lX*nX + lY*nY + lZ*nZ; //dot(l,n)
							if (aux>0){
								aux *= planeLightProperties[primitivePosition * 2];
								r += aux*planeColor[primitivePosition * 3] * lightColor[l * 3];
								g += aux*planeColor[primitivePosition * 3 + 1] * lightColor[l * 3 + 1];
								b += aux*planeColor[primitivePosition * 3 + 2] * lightColor[l * 3 + 2];
							}
						}

						////Calculate specular component
						if (1.0f - planeLightProperties[primitivePosition * 2]>0){
							//r=l-2dot(l,n)n
							rX = lX - 2.0f*(lX*nX + lY*nY + lZ*nZ)*nX;
							rY = lY - 2.0f*(lX*nX + lY*nY + lZ*nZ)*nY;
							rZ = lZ - 2.0f*(lX*nX + lY*nY + lZ*nZ)*nZ;

							aux = vX*rX + vY*rY + vZ*rZ;//dot(v,r)
							if (aux>0){
								aux *= powf(aux, 20)*(1.0f - planeLightProperties[primitivePosition * 2]);
								r += aux*lightColor[l * 3];
								g += aux*lightColor[l * 3 + 1];
								b += aux*lightColor[l * 3 + 2];


							}
						}


						//Calculate reflection
						if (planeLightProperties[2 * primitivePosition + 1] > 0)
						{
							//r=v-2dot(v,n)n
							rX = vX - 2.0f*(vX*nX + vY*nY + vZ*nZ)*nX;
							rY = vY - 2.0f*(vX*nX + vY*nY + vZ*nZ)*nY;
							rZ = vZ - 2.0f*(vX*nX + vY*nY + vZ*nZ)*nZ;

							aux = planeLightProperties[2 * primitivePosition + 1];
							float rRef, gRef, bRef;
							newRay(intersectionPointX + EPSILON*rX, intersectionPointY + EPSILON*rY, intersectionPointZ + EPSILON*rZ, rX, rY, rZ, rRef, gRef, bRef);

							r += aux *rRef * planeColor[primitivePosition * 4];
							g += aux*gRef * planeColor[primitivePosition * 4 + 1];
							b += aux*bRef * planeColor[primitivePosition * 4 + 2];
						}

					}

					////Calculate shade
					shade = 0.5f;

					intersections(intersectionPointX + EPSILON*lX, intersectionPointY + EPSILON*lY, intersectionPointZ + EPSILON*lZ, lX, lY, lZ, &primitiveTypeR, &primitivePositionR, &distanceR);
					if (primitiveTypeR == LIGHT){
						shade = 1.0f;
					}

					r *= shade; 
					g *= shade;
					b *= shade;		



				}

			}
			else{
				//	isActive=false;
			}
		
			output[globalTidY*width + globalTidX].x = (int)min(256.0f*r, 255.0f);
			output[globalTidY*width + globalTidX].y = (int)min(256.0f*g, 255.0f);
			output[globalTidY*width + globalTidX].z = (int)min(256.0f*b, 255.0f);
			output[globalTidY*width + globalTidX].w = 255;


		}
	}
}

//Scene initialization
/*float sphereH[NSPHERES * 4] = {0.0f, -1.0f, 10.0f, 1.5f,
-3.5f, -1.0f, 10.0f, 1.5f,
3.5f, -1.0f, 10.0f, 1.5f,
0.5f, 0.0f, 8.0f, 0.5f };
float sphereColorH[NSPHERES * 4] = { 1.0f, 0.0f, 0.0f, 0.0f,
0.0f, 1.0f, 0.0f, 0.0f,
0.0f, 0.0f, 1.0f, 0.0f,
1.0f, 0.5f, 0.0f, 0.0f };
float sphereLightPropertiesH[NSPHERES * 2] = {0.6f, 0.1f,
0.9f, 0.1f,
0.1f, 1.0f,
0.8f, 0.1f };

float planeH[NPLANES * 4] = { 0.0f, 1.0f, 0.0f, 4.0f };
float planeColorH[NPLANES * 3] = { 0.65f, 0.65f, 0.65f };
float planeLightPropertiesH[NPLANES * 2] = { 0.5f, 0.5f };

float lightH[NLIGHTS * 4] = { 3.0f, 1.0f, 0.0f, 0.2f };
float lightColorH[NLIGHTS * 3] = { 0.95f, 0.95f, 0.95f };
*/

float sphereH[NSPHERES * 4] = { -5.5f, -2.4, 7.0f, 2.0f,
								0.0f, -2.4, 7.0f, 2.0f, 
								 5.5f, -2.4, 7.0f, 2.0f };
float sphereColorH[NSPHERES * 4] = { 0.9f, 0.2f, 0.2f, 0.0,
									0.2f, 0.9f, 0.2f, 0.0f,
									0.2f, 0.2f, 0.9f, 0.0f };
float sphereLightPropertiesH[NSPHERES * 2] = { 0.8f, 0.8f,
												0.1f, 1.0f,
												0.2f, 0.8f };

float planeH[NPLANES * 4] = { 0.0f, 1.0f, 0.0f, 4.4f };
float planeColorH[NPLANES * 3] = { 0.4f, 0.4f, 0.4f };
float planeLightPropertiesH[NPLANES * 2] = { 0.5f, 0.8f };

float lightH[NLIGHTS * 4] = { 0.0f, 5.0f, 5.0f, 0.1f,
							2.0f, 5.0f, 1.0f, 0.1f };
float lightColorH[NLIGHTS * 3] = { 0.6f, 0.6f, 0.6f,
									0.9f, 0.9f, 0.9f };





static void draw_func(void) {
	// we pass zero as the last parameter, because out bufferObj is now
	// the source, and the field switches from being a pointer to a
	// bitmap to now mean an offset into a bitmap object
	glDrawPixels(WIDTH, HEIGHT, GL_RGBA, GL_UNSIGNED_BYTE, 0);
}

static void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods)
{
	if (key == GLFW_KEY_RIGHT && action == GLFW_PRESS)
	{
		lightH[0] += 0.5f;
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(light), lightH, NLIGHTS * 4 * sizeof(float)));
	}
	else if (key == GLFW_KEY_LEFT && action == GLFW_PRESS)
	{
		lightH[0] -= 0.5f;
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(light), lightH, NLIGHTS * 4 * sizeof(float)));
	}
	else if (key == GLFW_KEY_UP && action == GLFW_PRESS)
	{
		lightH[1] += 0.5f;
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(light), lightH, NLIGHTS * 4 * sizeof(float)));
	}
	else if (key == GLFW_KEY_DOWN && action == GLFW_PRESS)
	{
		lightH[1] -= 0.5f;
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(light), lightH, NLIGHTS * 4 * sizeof(float)));
	}
}


int main(int argc, char **argv) {
	

	//copy scene data to constant memory on GPU
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(sphere), sphereH, NSPHERES * 4 * sizeof(float)));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(sphereColor), sphereColorH, NSPHERES * 4 * sizeof(float)));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(sphereLightProperties), sphereLightPropertiesH, NSPHERES * 2 * sizeof(float)));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(plane), planeH, NPLANES * 4 * sizeof(float)));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(planeColor), planeColorH, NPLANES * 3 * sizeof(float)));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(planeLightProperties), planeLightPropertiesH, NPLANES * 2 * sizeof(float)));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(light), lightH, NLIGHTS * 4 * sizeof(float)));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(lightColor), lightColorH, NLIGHTS * 3 * sizeof(float)));
	


	
	hipDeviceProp_t  prop;
	int dev;



	memset(&prop, 0, sizeof(hipDeviceProp_t));
	prop.major = 1;
	prop.minor = 0;
	HANDLE_ERROR(hipChooseDevice(&dev, &prop));

	// tell CUDA which dev we will be using for graphic interop
	// from the programming guide:  Interoperability with OpenGL
	//     requires that the CUDA device be specified by
	//     cudaGLSetGLDevice() before any other runtime calls.

	//HANDLE_ERROR(cudaGLSetGLDevice(dev));

	GLFWwindow* window;

	// Initialise GLFW
	if (!glfwInit())
	{
		fprintf(stderr, "Failed to initialize GLFW\n");
		return -1;
	}


	// Open a window and create its OpenGL context
	window = glfwCreateWindow(WIDTH, HEIGHT, "bitmap", NULL, NULL);
	if (window == NULL){
		fprintf(stderr, "Failed to open GLFW window.\n");
		glfwTerminate();
		return -1;
	}

	glfwMakeContextCurrent(window);

	//glfwSwapInterval(1);

	// Initialize GLEW
	glewExperimental = true; // Needed for core profile
	if (glewInit() != GLEW_OK) {
		fprintf(stderr, "Failed to initialize GLEW\n");
		return -1;
	}



	/* Make the window's context current */
	glfwMakeContextCurrent(window);
	//leer teclado 
	glfwSetKeyCallback(window, key_callback);

	// the first three are standard OpenGL, the 4th is the CUDA reg 
	// of the bitmap these calls exist starting in OpenGL 1.5
	glGenBuffers(1, &bufferObj);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, bufferObj);
	glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, WIDTH * HEIGHT * 4,
		NULL, GL_DYNAMIC_DRAW_ARB);

	HANDLE_ERROR(
		hipGraphicsGLRegisterBuffer(&resource,
		bufferObj,
		cudaGraphicsMapFlagsNone));
	/* Loop GLFW until the user closes the window */
	while (!glfwWindowShouldClose(window))
	{
		
		// do work with the memory dst being on the GPU, gotten via mapping
		HANDLE_ERROR(hipGraphicsMapResources(1, &resource, NULL));
		uchar4* devPtr;
		size_t  size;
		HANDLE_ERROR(
			hipGraphicsResourceGetMappedPointer((void**)&devPtr,
			&size,
			resource));

		rayTrace <<<dim3(25, 20), dim3(32, 32) >>>(0, 0, -5, -4.0f, -3.0f, WIDTH, HEIGHT, devPtr);
		/* Render here */
		
		HANDLE_ERROR(hipGraphicsUnmapResources(1, &resource, NULL));
		draw_func();
		

		/* Swap front and back buffers */
		glfwSwapBuffers(window);

		/* Poll for and process events */
		glfwPollEvents();
	}

	glfwTerminate();
	return 0;

}
